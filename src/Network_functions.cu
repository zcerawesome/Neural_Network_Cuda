#include "hip/hip_runtime.h"
#include "Network_functions.cuh"

Dim2 blocks_threads(int size, int threads)
{
    int thread_amount = threads;
    int blocks = (size + thread_amount - 1) / thread_amount;
    return {blocks, thread_amount};
}

__global__ void normal_sample_value(float* result, float* rand_floats, int size, float mu, float sigma)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if(idx >= size)
        return;
    float z0 = sqrt(-2.0 * log(rand_floats[idx])) * cos(2.0 * M_PI * rand_floats[idx + size]);
    result[idx] = z0 * sigma + mu;
}

__global__ void generate_kernel(float* result, int size, unsigned long int seed)
{
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    if(id >= size)
        return;
    hiprandState state;
    hiprand_init(seed, id, 0, &state);
    result[id] = hiprand_uniform(&state) - 0.5;
}

__global__ void relu(const float* a, float* dest, int size)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if(idx < size)
        dest[idx] = a[idx] > 0 ? a[idx]: 0;
}

__global__ void relu_derive(const float* a, float* dest, int size)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if(idx < size)
        dest[idx] = a[idx] > 0;
}

__global__ void softmax(const float* a, float* dest, Dim2 a_dim)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if(idx < a_dim.col)
    {
        float sum = 0;
        for(int i = 0; i < a_dim.row; i++)
        {
            dest[i * a_dim.col + idx] = exp(a[i * a_dim.col + idx]);
            sum += dest[i * a_dim.col + idx];
        }
        for(int i = 0; i < a_dim.row; i++)
            dest[i * a_dim.col + idx] /= sum;
    }
}

__global__ void sigmoid(const float* a, float* dest, int size)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if(idx < size)
        dest[idx] = 1 / (1 + exp(-a[idx]));
}

__global__ void one_hot_encode(const float* a, float* dest, Dim2 a_dim)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if(idx < a_dim.col)
    {
        for(int i = 0; i < a_dim.row; i++)
        {
            dest[i * a_dim.col + idx] = 0;
        }
        dest[(int)(a[idx]) * a_dim.col + idx] = 1.0f;
    }
}

void randomize_matrix(matrice_gpu<float>& inp)
{
    int threads = 256;
    int blocks = (inp.size() + threads - 1) / threads;

    generate_kernel<<<blocks, threads>>>(inp.matrix, inp.size(), time(0));
    hipDeviceSynchronize();
}

void random_sample(matrice_gpu<float>& inp, float mu, float sigma)
{
    Dim2 threads_block = blocks_threads(inp.size(), 256);
    matrice_gpu<float> rand_data(1, inp.size() * 2);
    randomize_matrix(rand_data);
    rand_data = rand_data + 0.5;
    normal_sample_value<<<threads_block[0], threads_block[1]>>>(inp.matrix, rand_data.matrix, inp.size(), mu, sigma);
    hipDeviceSynchronize();
}

matrice_gpu<float> ReLU(matrice_gpu<float>& inp)
{
    matrice_gpu<float> temp(inp.numRows(), inp.numCols());
    int threads = 256;
    int blocks = (inp.size() + threads - 1) / threads;
    relu<<<blocks, threads>>>(inp.matrix, temp.matrix, inp.size());
    hipDeviceSynchronize();
    return temp;
}

matrice_gpu<float> ReLU_derive(matrice_gpu<float>& inp)
{
    Dim2 threads_block = blocks_threads(inp.size(), 256);
    matrice_gpu<float> temp(inp.numRows(), inp.numCols());
    relu_derive<<<threads_block[0], threads_block[1]>>>(inp.matrix, temp.matrix, inp.size());
    hipDeviceSynchronize();
    return temp;
}

matrice_gpu<float> softmax(matrice_gpu<float>& inp)
{
    Dim2 threads_block = blocks_threads(inp.numCols(), 256);
    matrice_gpu<float> temp(inp.numRows(), inp.numCols());
    softmax<<<threads_block[0], threads_block[1]>>>(inp.matrix, temp.matrix, {inp.numRows(), inp.numCols()});
    hipDeviceSynchronize();
    return temp;
}

matrice_gpu<float> sigmoid(matrice_gpu<float>& inp)
{
    Dim2 threads_block = blocks_threads(inp.size(), 256);
    matrice_gpu<float> temp(inp.numRows(), inp.numCols());
    sigmoid<<<threads_block[0], threads_block[1]>>>(inp.matrix, temp.matrix, inp.size());
    hipDeviceSynchronize();
    return temp;
}

matrice_gpu<float> one_hot_encode(matrice_gpu<float>& y, float max)
{
    matrice_gpu<float> temp(max + 1, y.numCols());
    Dim2 threads_block = blocks_threads(y.numCols(), 256);

    one_hot_encode<<<threads_block[0], threads_block[1]>>>(y.matrix, temp.matrix, {temp.numRows(), temp.numCols()});    
    hipDeviceSynchronize();
    return temp;
}