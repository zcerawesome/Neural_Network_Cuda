#include "hip/hip_runtime.h"
#include "matrice_helper.cuh"

void set_data(void* a, void* b, void* c, int n)
{
    hipMalloc((void**)a, n);
    hipMalloc((void**)b, n);
    hipMalloc((void**)c, n);
}

void free_data(void* a, void* b, void* c)
{
    hipFree(a);
    hipFree(b);
    hipFree(c);
}

template <typename T>
__global__ void dot(const T* a, const T* b, T* dest, int m, int k, int n)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    if(idx < m && idy < n)
    {
        T sum = 0;
        for(int i = 0; i < k; i++)
            sum += a[idx * k + i] * b[i * n + idy];
        dest[idx * n + idy] = sum;
    }

}

template <typename T>
__global__ void General_operation(const T* a, const T* b, T* dest, Dim2 a_dim, Dim2 b_dim, Operations op)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    if(idx < a_dim.row && idy < a_dim.col)
    {
        int regular_index = idx * a_dim.col + idy; 
        bool row_equal = a_dim.row == b_dim.row;
        bool col_equal = a_dim.col == b_dim.col;
        bool regular_operation = row_equal && col_equal;

        int b_index = (regular_operation) * regular_index + (!regular_operation) * (row_equal * idx + col_equal * idy);
        switch(op)
        {
            case Add:
                dest[regular_index] = a[regular_index] + b[b_index];
                break;
            case Subtract:
                dest[regular_index] = a[regular_index] - b[b_index];
                break;
            case Multiply:
                dest[regular_index] = a[regular_index] * b[b_index];
                break;
            case Division:
                dest[regular_index] = a[regular_index] / b[b_index];
        }
    }
}

template <typename T>
__global__ void General_scalar_operation(const T* a, T scalar, T* dest, int n, Operations op)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if(idx >= n)
        return;
    switch(op)
    {
        case Add:
            dest[idx] = a[idx] + scalar;
            break;
        case Subtract:
            dest[idx] = a[idx] - scalar;
            break;
        case Multiply:
            dest[idx] = a[idx] * scalar;
            break;
        case Division:
            dest[idx] = a[idx] / scalar;
    }
}



template <typename T>
void General_scalar_helper(const T* a, T scalar, T* dest, int n, Operations op)
{
    int threadsPerBlock = 256;
    int blocks = (n + threadsPerBlock - 1) / threadsPerBlock;
    T* cudaA, *cudaDest;

    hipMalloc(&cudaA, n * sizeof(T));
    hipMalloc(&cudaDest, n * sizeof(T));
    hipMemcpy(cudaA, a, n * sizeof(T), hipMemcpyHostToDevice);

    General_scalar_operation<T><<<blocks, threadsPerBlock>>>(cudaA, scalar, cudaDest, n, op);
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
        std::cerr << "Cuda Scalar Error: " << hipGetErrorString(error) << std::endl;

    hipMemcpy(dest, cudaDest, n * sizeof(T), hipMemcpyDeviceToHost);
    hipFree(cudaA);
    hipFree(cudaDest);
}

template <typename T>
void General_operation_helper(const T* a, const T* b, T* dest, Operations op, Dim2 a_dim, Dim2 b_dim)
{
    int threadX = 16;
    int blockX = (a_dim.row + threadX - 1) / threadX;

    int threadY = 16;
    int blockY = (a_dim.col + threadY - 1) / threadY;


    T* cudaA;
    T* cudaB;
    T* cudaDest;

    hipMalloc(&cudaA, a_dim.row * a_dim.col * sizeof(T));
    hipMalloc(&cudaB, b_dim.row * b_dim.col * sizeof(T));
    hipMalloc(&cudaDest, a_dim.row * a_dim.col * sizeof(T));

    hipMemcpy(cudaA, a, a_dim.row * a_dim.col * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(cudaB, b, b_dim.row * b_dim.col * sizeof(T), hipMemcpyHostToDevice);

    dim3 threads(threadX, threadY);
    dim3 Blocks(blockX, blockY);
    General_operation<T><<<Blocks, threads>>>(cudaA, cudaB, cudaDest, a_dim, b_dim, op);
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
        std::cerr << "Cuda General Error: " << hipGetErrorString(error) << std::endl;

    hipMemcpy(dest, cudaDest, a_dim.row * a_dim.col * sizeof(T), hipMemcpyDeviceToHost);
    free_data(cudaA, cudaB, cudaDest);
}


template <typename T>
void dot_product(const T* a, const T* b, T* dest, int m, int k, int n)
{
    int threadX = 16;
    int blockX = (m + threadX - 1) / threadX;

    int threadY = 16;
    int blockY = (n + threadY - 1) / threadY;
 
    T* cudaA, *cudaB, *cudaDest;
    
    hipMalloc(&cudaA, m * k * sizeof(T));
    hipMalloc(&cudaB, n * k * sizeof(T));
    hipMalloc(&cudaDest, m * n * sizeof(T));

    hipMemcpy(cudaA, a, m * k * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(cudaB, b, n * k * sizeof(T), hipMemcpyHostToDevice);
        
    dim3 threads(threadX, threadY);
    dim3 Blocks(blockX, blockY);
    dot<<<Blocks, threads>>>(cudaA, cudaB, cudaDest, m, k, n);
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
        std::cerr << "Cuda Dot Error: " << hipGetErrorString(error) << std::endl;

    hipMemcpy(dest, cudaDest, m * n * sizeof(T), hipMemcpyDeviceToHost);
    free_data(cudaA, cudaB, cudaDest);

}

template void General_operation_helper(const int* , const int*, int*, Operations, Dim2, Dim2);
template void General_operation_helper(const float* , const float*, float*, Operations, Dim2, Dim2);

template void General_scalar_helper(const int* a, int scalar, int* dest, int n, Operations);
template void General_scalar_helper(const float* a, float scalar, float* dest, int n, Operations);

template void dot_product(const int* , const int* , int*, int, int, int);
template void dot_product(const float* , const float* , float*, int, int, int);
