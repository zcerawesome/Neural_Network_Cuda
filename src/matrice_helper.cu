#include "hip/hip_runtime.h"
#include "matrice_helper.cuh"

void set_data(void* a, void* b, void* c, int n)
{
    hipMalloc((void**)a, n);
    hipMalloc((void**)b, n);
    hipMalloc((void**)c, n);
}

void free_data(void* a, void* b, void* c)
{
    hipFree(a);
    hipFree(b);
    hipFree(c);
}


template <typename T>
__global__ void General_operation(const T* a, const T* b, T* dest, int n, Operations op)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if(idx >= n)
        return;
    switch(op)
    {
        case Add:
            dest[idx] = a[idx] + b[idx];
            break;
        case Subtract:
            dest[idx] = a[idx] - b[idx];
            break;
        case Multiply:
            dest[idx] = a[idx] * b[idx];
            break;
        case Division:
            dest[idx] = a[idx] / b[idx];
    }
}

template <typename T>
__global__ void General_scalar_operation(const T* a, T scalar, T* dest, int n, Operations op)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if(idx >= n)
        return;
    switch(op)
    {
        case Add:
            dest[idx] = a[idx] + scalar;
            break;
        case Subtract:
            dest[idx] = a[idx] - scalar;
            break;
        case Multiply:
            dest[idx] = a[idx] * scalar;
            break;
        case Division:
            dest[idx] = a[idx] / scalar;
    }
}



template <typename T>
void General_scalar_helper(const T* a, T scalar, T* dest, int n, Operations op)
{
    int threadsPerBlock = 256;
    int blocks = (n + threadsPerBlock - 1) / threadsPerBlock;
    T* cudaA, *cudaDest;

    hipMalloc(&cudaA, n * sizeof(T));
    hipMalloc(&cudaDest, n * sizeof(T));
    hipMemcpy(cudaA, a, n * sizeof(T), hipMemcpyHostToDevice);

    General_scalar_operation<T><<<blocks, threadsPerBlock>>>(cudaA, scalar, cudaDest, n, op);
    hipDeviceSynchronize();

    hipMemcpy(dest, cudaDest, n * sizeof(T), hipMemcpyDeviceToHost);
    hipFree(cudaA);
    hipFree(cudaDest);
}

template <typename T>
void General_operation_helper(const T* a, const T* b, T* dest, int n, Operations op)
{
    int threadsPerBlock = 256;
    int blocks = (n + threadsPerBlock -1) / threadsPerBlock;

    T* cudaA;
    T* cudaB;
    T* cudaDest;

    set_data(&cudaA, &cudaB, &cudaDest, n * sizeof(T));

    hipMemcpy(cudaA, a, n * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(cudaB, b, n * sizeof(T), hipMemcpyHostToDevice);

    General_operation<T><<<blocks, threadsPerBlock>>>(cudaA, cudaB, cudaDest, n, op);
    hipDeviceSynchronize();
    hipMemcpy(dest, cudaDest, n * sizeof(T), hipMemcpyDeviceToHost);
    free_data(cudaA, cudaB, cudaDest);
}

template <typename T>
__global__ void dot(const T* a, const T* b, T* dest, int m, int k, int n)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    if(idx < m && idy < n)
    {
        T sum = 0;
        for(int i = 0; i < k; i++)
            sum += a[idx * k + i] * b[i * n + idy];
        dest[idx * n + idy] = sum;
    }

}

template <typename T>
void dot_product(const T* a, const T* b, T* dest, int m, int k, int n)
{
    int threadX = 16;
    int blockX = (m + threadX - 1) / threadX;

    int threadY = 16;
    int blockY = (n + threadY - 1) / threadY;
 
    T* cudaA, *cudaB, *cudaDest;
    
    hipMalloc(&cudaA, m * k * sizeof(T));
    hipMalloc(&cudaB, n * k * sizeof(T));
    hipMalloc(&cudaDest, m * n * sizeof(T));

    hipMemcpy(cudaA, a, m * k * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(cudaB, b, n * k * sizeof(T), hipMemcpyHostToDevice);
        
    dim3 threads(threadX, threadY);
    dim3 Blocks(blockX, blockY);
    dot<<<Blocks, threads>>>(cudaA, cudaB, cudaDest, m, k, n);
    hipDeviceSynchronize();

    hipMemcpy(dest, cudaDest, m * n * sizeof(T), hipMemcpyDeviceToHost);
    free_data(cudaA, cudaB, cudaDest);

}

template <typename T>
__global__ void sumting(T* a, int size, int iteration)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if(idx > size || idx % (int)(pow(2,iteration+1)))
        return;
    a[idx] = a[idx] + (idx + pow(2, iteration) < size) * a[idx + (int)pow(2, iteration)];
}

template <typename T>
void sum_cuda(const T* a, T* dest, int size)
{
    T* cudaA;
    hipMalloc(&cudaA, size * sizeof(T));
    hipMemcpy(cudaA, a, size * sizeof(T), hipMemcpyHostToDevice);

    int iterations = (int)(log2(size)) + 1;
    if(log2(size) == iterations-1)
        iterations--;
    int threads = 256;
    int blocks = (size + threads - 1) / threads;
    for(int i = 0; i < iterations; i++)
    {
        sumting<T><<<blocks, threads>>>(cudaA, size, i);
        hipDeviceSynchronize();
    }
    // sum_helper<<<1, 1>>>(cudaA, cudaDest, size, iterations);

    hipMemcpy(dest, cudaA, sizeof(T), hipMemcpyDeviceToHost);
    hipFree(cudaA);

}

template void General_operation_helper(const int* , const int*, int*, int, Operations);
template void General_operation_helper(const float* , const float*, float*, int, Operations);

template void General_scalar_helper(const int* a, int scalar, int* dest, int n, Operations);
template void General_scalar_helper(const float* a, float scalar, float* dest, int n, Operations);

template void dot_product(const int* , const int* , int*, int, int, int);
template void dot_product(const float* , const float* , float*, int, int, int);

template void sum_cuda(const int*, int*, int);
template void sum_cuda(const float*, float*, int);