#include "hip/hip_runtime.h"
#include "matrice_helper.cuh"

void set_data(void* a, void* b, void* c, int n)
{
    hipMalloc((void**)a, n);
    hipMalloc((void**)b, n);
    hipMalloc((void**)c, n);
}

void free_data(void* a, void* b, void* c)
{
    hipFree(a);
    hipFree(b);
    hipFree(c);
}

template <typename T>
__global__ void dot(const T* a, const T* b, T* dest, int m, int k, int n)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    if(idx < m && idy < n)
    {
        T sum = 0;
        for(int i = 0; i < k; i++)
            sum += a[idx * k + i] * b[i * n + idy];
        dest[idx * n + idy] = sum;
    }

}

template <typename T>
__global__ void General_operation(const T* a, const T* b, T* dest, Dim2 a_dim, Dim2 b_dim, Operations op)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    if(idx < a_dim.row && idy < a_dim.col)
    {
        int regular_index = idx * a_dim.col + idy; 
        bool row_equal = a_dim.row == b_dim.row;
        bool col_equal = a_dim.col == b_dim.col;
        bool regular_operation = row_equal && col_equal;

        int b_index = (regular_operation) * regular_index + (!regular_operation) * (row_equal * idx + col_equal * idy);
        switch(op)
        {
            case Add:
                dest[regular_index] = a[regular_index] + b[b_index];
                break;
            case Subtract:
                dest[regular_index] = a[regular_index] - b[b_index];
                break;
            case Multiply:
                dest[regular_index] = a[regular_index] * b[b_index];
                break;
            case Division:
                dest[regular_index] = a[regular_index] / b[b_index];
        }
    }
}

template <typename T>
__global__ void General_scalar_operation(const T* a, T scalar, T* dest, int n, Operations op)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if(idx >= n)
        return;
    switch(op)
    {
        case Add:
            dest[idx] = a[idx] + scalar;
            break;
        case Subtract:
            dest[idx] = a[idx] - scalar;
            break;
        case Multiply:
            dest[idx] = a[idx] * scalar;
            break;
        case Division:
            dest[idx] = a[idx] / scalar;
    }
}

template <typename T>
void General_scalar_helper(const T* a, T scalar, T* dest, int n, Operations op)
{
    int threadsPerBlock = 256;
    int blocks = (n + threadsPerBlock - 1) / threadsPerBlock;

    General_scalar_operation<<<blocks, threadsPerBlock>>>(a, scalar, dest, n, op);
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
        std::cerr << "Cuda Scalar Error: " << n << " "<< hipGetErrorString(error) << std::endl;

}

template <typename T>
void General_operation_helper(const T* a, const T* b, T* dest, Operations op, Dim2 a_dim, Dim2 b_dim)
{
    int threadX = 16;
    int blockX = (a_dim.row + threadX - 1) / threadX;

    int threadY = 16;
    int blockY = (a_dim.col + threadY - 1) / threadY;


    dim3 threads(threadX, threadY);
    dim3 Blocks(blockX, blockY);
    General_operation<T><<<Blocks, threads>>>(a, b, dest, a_dim, b_dim, op);
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cerr << "Cuda General Error: " << hipGetErrorString(error) << std::endl;
        std::cerr << a_dim[0] << " " << a_dim[1] << "  " << b_dim[0] << " " << b_dim[1] << std::endl;
    }

}


template <typename T>
inline void dot_product(const T* a, const T* b, T* dest, int m, int k, int n)
{
    int threadX = 16;
    int blockX = (m + threadX - 1) / threadX;

    int threadY = 16;
    int blockY = (n + threadY - 1) / threadY;
 
    
            
    dim3 threads(threadX, threadY);
    dim3 Blocks(blockX, blockY);
    dot<<<Blocks, threads>>>(a, b, dest, m, k, n);
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cerr << "Cuda Dot Error: " << hipGetErrorString(error) << std::endl;
        std::cerr << m << " " << k << "  " << k << " " << n << std::endl;
    }

}

template <typename T>
__global__ void transpose_Cuda(const T* a, T* dest, Dim2 a_dim)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    if(idx < a_dim.row && idy < a_dim.col)
        dest[idy * a_dim.row + idx] = a[idx * a_dim.col + idy];
}

template <typename T>
void transpose_GPU(const T* a, T* dest, Dim2 a_dim)
{
    int threadX = 16;
    int blockX = (a_dim.row + threadX - 1) / threadX;

    int threadY = 16;
    int blockY = (a_dim.col + threadY - 1) / threadY;
    dim3 threads(threadX, threadY);
    dim3 Blocks(blockX, blockY);

    transpose_Cuda<<<Blocks, threads>>>(a, dest, a_dim);
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
        std::cerr << "Cuda Transpose Error: " << hipGetErrorString(error) << std::endl;
}

template void General_operation_helper(const int* , const int*, int*, Operations, Dim2, Dim2);
template void General_operation_helper(const float* , const float*, float*, Operations, Dim2, Dim2);

template void General_scalar_helper(const int* a, int scalar, int* dest, int n, Operations);
template void General_scalar_helper(const float* a, float scalar, float* dest, int n, Operations);

template void dot_product(const int* , const int* , int*, int, int, int);
template void dot_product(const float* , const float* , float*, int, int, int);

template void transpose_GPU(const float* , float* , Dim2);
template void transpose_GPU(const int* , int* , Dim2);
